#include "hip/hip_runtime.h"
//
// Created by emichel on 05/07/24.
//

#include "difx_gpu_header.h"

__constant__ float kLevels_2bit[4];

void init_2bitLevels() {
    static const float HiMag = 3.3359;  // Optimal value
    const float lut4level[4] = {-HiMag, -1.0, 1.0, HiMag};
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(kLevels_2bit), lut4level, 4 * sizeof(float), 0, hipMemcpyHostToDevice));
}

// Rotate inplace a complex number by theta (radians)
__device__ static __inline__ void cuRotatePhase(COMPLEX &x, float theta) {
    float cs, sn;
    sincosf(theta, &sn, &cs);

#ifdef USEHALF
    float2 y = __half22float2(x);
  float px = y.x * cs - y.y * sn;
  float py = y.x * sn + y.y * cs;
#else
    float px = x.x * cs - x.y * sn;
    float py = x.x * sn + x.y * cs;
#endif
    x = MAKECOMPLEX(px, py);
    return;
}

// Rotate a complex number by theta (radians)
__device__ static __inline__ void cuRotatePhase3(float x, COMPLEX &y, float sinA, float cosA) {
    y = MAKECOMPLEX(x * cosA, x * sinA);
    return;
}

// Rotate a complex number by theta (radians)
__device__ static __inline__ void cuRotatePhase4(hipComplex x, COMPLEX &y, float sinA, float cosA) {
    y = MAKECOMPLEX(x.x * cosA - x.y * sinA, x.x * sinA + x.y * cosA);
    return;
}


void loadCfg(int NANT, int *nbit, int *nPol, int *iscomplex, int *numchannels, int *numantennas, double *lo,
             double *bandwidth, int *numffts, char **antFiles, double *delays, int *samplegranularity,
             int *nbaselines, int *fftsamples, int *subintsamples, double *sampletime, int *subintbytes) {
    char *configFileName = "./dat/test4_tested.conf";
    FILE *fconfig = fopen(configFileName, "r");

    for (int i = 0; i < MAX_ANTENNAS; i++) {
        antFiles[i] = (char *)malloc(MAX_ANTENNA_FILE_NAME * sizeof(char));
    }

    if (fconfig == NULL) {
        printf("Error Opening File %s\n", configFileName);
        return;
    }

    char line[1024];
    int antToRead = 0;
    int iant = 0;

    // Set some defaults
    *nPol = 2;
    *iscomplex = 0;
    *nbit = 2;

    // Read the config file
    while (fgets(line, 1024, fconfig)) {
        char keyword[1024];
        char *token = strtok(line, " \n");
        if (!token) {
            fprintf(stderr, "Error: Could not parse \"%s\"\n", line);
            exit(1);
        }
        strcpy(keyword, token);
        if (antToRead) {
            char thisfile[1024];
            sscanf(keyword, "%s", thisfile);
            //strcpy(antenna[iant], keyword);
            token = strtok(NULL, " \n");
            strcpy(antFiles[iant], token);
            token = strtok(NULL, " \n");
            for (int i = 0; i < 3; i++) {
                if (token == NULL) {
                    fprintf(stderr, "Error: Insufficient data for antenna %d\n", iant);
                    exit(1);
                }
                sscanf(token, "%lf", &((delays)[4 * iant + i])); // TODO CHECK
                token = strtok(NULL, " \n");
            }
            if (token == NULL) {
                fprintf(stderr, "Error: Insufficient data for antenna %d\n", iant);
                exit(1);
            }
            sscanf(token, "%lf", &((delays)[4 * iant + 3])); // Error checking needed
            iant++;
            antToRead--;
        } else {
            if (strcasecmp(keyword, "COMPLEX") == 0) {
                token = strtok(NULL, " \n");
                if (token == NULL) {
                    fprintf(stderr, "Error: Insufficient data for COMPLEX\n");
                    exit(1);
                }
                sscanf(token, "%d", iscomplex); // Should error check
            } else if (strcasecmp(keyword, "NBIT") == 0) {
                token = strtok(NULL, " \n");
                if (token == NULL) {
                    fprintf(stderr, "Error: Insufficient data for NBIT\n");
                    exit(1);
                }
                sscanf(token, "%d", nbit); // Should error check
            } else if (strcasecmp(keyword, "NPOL") == 0) {
                token = strtok(NULL, " \n");
                if (token == NULL) {
                    fprintf(stderr, "Error: Insufficient data for NPOL\n");
                    exit(1);
                }
                sscanf(token, "%d", nPol); // Should error check
            } else if (strcasecmp(keyword, "NCHAN") == 0) {
                token = strtok(NULL, " \n");
                if (token == NULL) {
                    fprintf(stderr, "Error: Insufficient data for NCHAN\n");
                    exit(1);
                }
                sscanf(token, "%d", numchannels); // Should error check
            } else if (strcasecmp(keyword, "LO") == 0) {
                token = strtok(NULL, " \n");
                if (token == NULL) {
                    fprintf(stderr, "Error: Insufficient data for LO\n");
                    exit(1);
                }
                sscanf(token, "%lf", lo); // Should error check
            } else if (strcasecmp(keyword, "BANDWIDTH") == 0) {
                token = strtok(NULL, " \n");
                if (token == NULL) {
                    fprintf(stderr, "Error: Insufficient data for BANDWIDTH\n");
                    exit(1);
                }
                sscanf(token, "%lf", bandwidth); // Should error check
            } else if (strcasecmp(keyword, "NUMFFTS") == 0) {
                token = strtok(NULL, " \n");
                if (token == NULL) {
                    fprintf(stderr, "Error: Insufficient data for NUMFFTS\n");
                    exit(1);
                }
                sscanf(token, "%d", numffts); // Should error check
            } else if (strcasecmp(keyword, "NANT") == 0) {
                token = strtok(NULL, " \n");
                if (token == NULL) {
                    fprintf(stderr, "Error: Insufficient data for NANT\n");
                    exit(1);
                }
                sscanf(token, "%d", numantennas); // Should error check
                antToRead = *numantennas;
                iant = 0;
            } else {
                fprintf(stderr, "Error: Unknown keyword \"%s\"\n", keyword);
            }
        }
    }
    fclose(fconfig);

    // Check that the number of FFTs is a valid number
    if (*numffts % 8) {
        printf("Error: numffts must be divisible by 8");
        exit(1);
    }

//    int *samplegranularity,
//    int *nbaselines, int *fftsamples, int *subintsamples, double *sampletime, int *subintbytes) {

    *samplegranularity = 8 / (*nbit * *nPol);
    if (*samplegranularity < 1) {
        *samplegranularity = 1;
    }
    *nbaselines = *numantennas * (*numantennas - 1) / 2;

    int cfactor;
    if (*iscomplex) {
        cfactor = 1;
    } else {
        cfactor = 2; // If real data FFT size twice size of number of frequecy channels
    }

    *fftsamples = *numchannels * cfactor;
    *subintsamples = *numffts * *fftsamples;  // Number of time samples - need to factor # channels (pols) also
    printf("Subintsamples = %d\n", *subintsamples);
    *sampletime = 1.0 / *bandwidth;

    if (!(*iscomplex)) *sampletime /= 2.0;

    *subintbytes = *numchannels * cfactor * (*numffts + 1) * *nbit / 8 * *nPol;
}

void
deviceSetup(int *fftsamples, int *nPol, int *numantennas, int *numffts, int *nbit, int *subintsamples, int *iscomplex,
            int *numchannels, int *subintbytes, hipfftHandle *plan) {

    if (hipfftPlan1d(plan, *fftsamples, HIPFFT_C2C, *nPol * *numantennas * *numffts) != HIPFFT_SUCCESS) {
        printf("CUFFT error: Plan creation failed");
        exit(0);
    }
}


void readData(int NBIT, int NPOL, int CFACTOR, int NFFTS, int NANT, int NCHAN, int *subintbytes, FILE **antStream,
              int *numantennas, int8_t *inputData) {

    printf("Size subintbytes %d \n", *subintbytes);

    printf("Size antStream : %d \n", *numantennas);
    for (int i = 0; i < *numantennas; i++) {
        size_t bytes_read = fread(&inputData[i * *subintbytes], 1, *subintbytes, antStream[i]);
        if (bytes_read != *subintbytes) {
            if (feof(antStream[i])) {
                printf("2"); // End of file reached
            } else if (ferror(antStream[i])) {
                perror("Error: Problem reading data");
                printf("1"); // Error reading file
            }
        }
    }
    printf("Read Data Success \n"); // Successful read
}

void openAntFiles(int NANT, int *numantennas, char **antFiles, FILE **antStream) {
    for (int i = 0; i < NANT; i++) {
        antStream[i] = fopen(antFiles[i], "rb");
        if (antStream[i] == NULL) {
            fprintf(stderr, "Problem with file %s - does it exist?\n", antFiles[i]);
        }
    }
}

__global__ void
calculateDelaysAndPhasesGPU(int NANT, int NFFTS, double *delays, double *lo, double *sampletime, int *fftsamples,
                            int *numchannels, int *samplegranularity, float *rotationPhaseInfo, int *sampleShifts,
                            int *fractionalSampleDelays, uint8_t *packedData, uint8_t *inputData) {
    size_t ifft = threadIdx.x + blockIdx.x * blockDim.x;
    size_t iant = blockIdx.y;
    int numffts = blockDim.x * gridDim.x;
    double meandelay, deltadelay, netdelaysamples_f, startphase;
    double d0, d1, d2, a, b;
    double *interpolator = &(delays[iant * 4]);
    double filestartoffset = delays[iant * 4 + 3];
    float fractionaldelay;
    int netdelaysamples;

    // evaluate the delay for the given FFT of the given antenna

    // calculate values at the beginning, middle, and end of this FFT
    d0 = interpolator[0] * ifft * ifft + interpolator[1] * ifft + interpolator[2];
    d1 = interpolator[0] * (ifft + 0.5) * (ifft + 0.5) + interpolator[1] * (ifft + 0.5) + interpolator[2];
    d2 = interpolator[0] * (ifft + 1.0) * (ifft + 1.0) + interpolator[1] * (ifft + 1.0) + interpolator[2];

    // use these to calculate a linear interpolator across the FFT, as well as a mean value
    a = d2 - d0; //this is the delay gradient across this FFT
    b = d0 + (d1 - (a * 0.5 + d0)) / 3.0; //this is the delay at the start of the FFT
    meandelay = a * 0.5 + b; //this is the delay in the middle of the FFT
    deltadelay = a / *fftsamples; // this is the change in delay per sample across this FFT window

    netdelaysamples_f = (meandelay - filestartoffset) / *sampletime;
    netdelaysamples = __double2int_rn(netdelaysamples_f / *samplegranularity) * *samplegranularity;

    // Save the integer number of sample shifts
    sampleShifts[iant * numffts + ifft] = netdelaysamples;

    // Save the fractional delay
    fractionaldelay = (float) (-(netdelaysamples_f - netdelaysamples) * 2 * M_PI /
                               *fftsamples);  // radians per FFT channel
    fractionalSampleDelays[iant * numffts + ifft] = fractionaldelay;

    // set the fringe rotation phase for the first sample of a given FFT of a given antenna
    startphase = b * *lo;
    rotationPhaseInfo[iant * numffts * 2 + ifft * 2] = (float) (startphase - int(startphase)) * 2 * M_PI;
    rotationPhaseInfo[iant * numffts * 2 + ifft * 2 + 1] = (float) (deltadelay * *lo) * 2 * M_PI;
}

__global__ void
unpackData(int NANT, int NFFTS, int CFACTOR, int NCHAN, int NPOL, int NBIT, int SUBINTSAMP, int8_t *packedData,
           float *rotationPhaseInfo, int *sampleShifts, int *fftsamples, COMPLEX *unpackedData) {
    if(threadIdx.x < *fftsamples/2 && blockIdx.x < NFFTS) {
        for (int i = 0; i < NANT; i++) {
            // static const float HiMag = 3.3359;  // Optimal value
            // const float levels_2bit[4] = {-HiMag, -1.0, 1.0, HiMag};
            const size_t isample = 2 * threadIdx.x;
            const size_t ifft = blockIdx.x;
            const size_t osample = isample + ifft * *fftsamples;
            int subintsamples = *fftsamples * NFFTS;

            // Try to Fix
            size_t idx = ((osample - sampleShifts[ifft + i * NFFTS]) /
                          2); // FIXME: may lead to memory access outside src[] bounds, see with 'cuda-memcheck ./benchmark_gxkernel'
            // And of try to fix

            int8_t src_i = packedData[idx + i * (NCHAN * CFACTOR * (NFFTS + 1) * NBIT / 8 *
                                                 NPOL)]; // Here I am just loading src into local memory to
            // reduce the number of reads from global memory

            // I have just changed the order of the writes made to dest
            // In theory this should reduce the number of write operations made
            // I have also implemented the use of constant memory for the levels_2bit
            // array

            float samp0 = kLevels_2bit[src_i & 0x3];
            float samp1 = kLevels_2bit[(src_i >> 4) & 0x3];
            float samp2 = kLevels_2bit[(src_i >> 2) & 0x3];
            float samp3 = kLevels_2bit[(src_i >> 6) & 0x3];

            // phase and slope for this FFT
            float p0 = rotationPhaseInfo[ifft * 2 + i * NFFTS * 2];
            float p1 = rotationPhaseInfo[ifft * 2 + 1 + i * NFFTS * 2];
            float theta0 = -p0 - isample * p1;
            float theta1 = -p0 - (isample + 1) * p1;

            float sinT0, cosT0, sinT1, cosT1;
            sincosf(theta0, &sinT0, &cosT0);
            sincosf(theta1, &sinT1, &cosT1);
            cuRotatePhase3(samp0, unpackedData[(osample + 2 * i * SUBINTSAMP)], sinT0, cosT0);
            cuRotatePhase3(samp1, unpackedData[(osample + 1 + 2 * i * SUBINTSAMP)], sinT1, cosT1);
            cuRotatePhase3(samp2, unpackedData[(subintsamples + osample + 2 * i * SUBINTSAMP)], sinT0, cosT0);
            cuRotatePhase3(samp3, unpackedData[(subintsamples + osample + 1 + 2 * i * SUBINTSAMP)], sinT1, cosT1);
        }
    }
}

void
doFFT(int SUBINTSAMP, int NANT, int NPOL, COMPLEX *unpackedData, hipfftHandle *plan, COMPLEX *channelisedData) {
    if (hipfftExecC2C(*plan, unpackedData, channelisedData, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
        printf("CUFFT error: ExecC2C Forward failed");
    }
}

__global__ void fracSampleCorrection_old(int NANT, int NFFTS, int SUBINTSAMP, int NPOL, COMPLEX *channelisedData,
                                     float *fractionalSampleDelays, int *numchannels, int *fftsamples, int *numffts,
                                     int *subintsamples, COMPLEX *channelisedData_out) {
    size_t ichan = threadIdx.x + blockIdx.x * blockDim.x;
    size_t ifft = blockIdx.y;
    size_t iant = blockIdx.z;

    // phase and slope for this FFT
    float dslope = fractionalSampleDelays[iant * *numffts + ifft];
    float theta = ichan * dslope;
    cuRotatePhase(channelisedData[sampIdx(iant, 0, ichan + ifft * *fftsamples, *subintsamples)], theta);
    cuRotatePhase(channelisedData[sampIdx(iant, 1, ichan + ifft * *fftsamples, *subintsamples)], theta);

    memcpy(&channelisedData_out[sampIdx(iant, 0, ichan + ifft * *fftsamples, *subintsamples)], &channelisedData[sampIdx(iant, 0, ichan + ifft * *fftsamples, *subintsamples)], sizeof(hipComplex));
    memcpy(&channelisedData_out[sampIdx(iant, 1, ichan + ifft * *fftsamples, *subintsamples)], &channelisedData[sampIdx(iant, 1, ichan + ifft * *fftsamples, *subintsamples)], sizeof(hipComplex));
}

__global__ void fracSampleCorrection(int NANT, int NFFTS, int SUBINTSAMP, int NPOL, COMPLEX *channelisedData,
                                     float *fractionalSampleDelays, int *numchannels, int *fftsamples, int *numffts,
                                     int *subintsamples, COMPLEX *channelisedData_out) {
    if(threadIdx.x < *numchannels && blockIdx.x < NFFTS*NANT) { // NUMCHAN et NUMFFTS*NUMANTENNAS
        size_t ichan = threadIdx.x;
        size_t combinedIndex = blockIdx.x;
        size_t ifft = combinedIndex % *numffts;
        size_t iant = combinedIndex / *numffts;

        // phase and slope for this FFT
        float dslope = fractionalSampleDelays[iant * *numffts + ifft];
        float theta = ichan * dslope;
        cuRotatePhase(channelisedData[sampIdx(iant, 0, ichan + ifft * *fftsamples, SUBINTSAMP)], theta);
        cuRotatePhase(channelisedData[sampIdx(iant, 1, ichan + ifft * *fftsamples, SUBINTSAMP)], theta);

        memcpy(&channelisedData_out[sampIdx(iant, 0, ichan + ifft * *fftsamples, *subintsamples)], &channelisedData[sampIdx(iant, 0, ichan + ifft * *fftsamples, *subintsamples)], sizeof(hipComplex));
        memcpy(&channelisedData_out[sampIdx(iant, 1, ichan + ifft * *fftsamples, *subintsamples)], &channelisedData[sampIdx(iant, 1, ichan + ifft * *fftsamples, *subintsamples)], sizeof(hipComplex));
    }
}


__global__ void
CCAH(int NPOL, int SUBINTSAMP, int NANT, int NBASELINE, int NCHAN, int PARAACCUM, COMPLEX *channelisedData,
     int *numantennas, int *numffts, int *numchannels, int *fftsamples, COMPLEX *baselineData) {
    if(blockIdx.x < (1+(NCHAN-1)/128)*NANT*NANT && threadIdx.x < 128) {
        int linear_idx = blockIdx.x * 128 + threadIdx.x;
        // Total number of pairs of antennas
        int total_pairs = NANT * (NANT - 1) / 2;
        int total_elements = total_pairs * NCHAN;

        if (linear_idx >= total_elements) return;

        // Calculate the channel index
        int t = linear_idx % NCHAN;

        // Calculate the pair index
        int pair_idx = linear_idx / NCHAN;

        // Calculate ant1 and ant2 from pair_idx
        int ant1 = 0;
        int ant2 = 0;
        int offset = 0;
        for (ant1 = 0; ant1 < NANT - 1; ++ant1) {
            int remaining_pairs = NANT - ant1 - 1;
            if (pair_idx < offset + remaining_pairs) {
                ant2 = ant1 + 1 + (pair_idx - offset);
                break;
            }
            offset += remaining_pairs;
        }

        // Index into output vector blocks: = (j-i-1) + n-1 + ... + n-i
        int b = ant1 * NANT - ant1 * (ant1 + 1) / 2 - ant1 + ant2 - 1;

        int s = *numffts * *fftsamples;

        const COMPLEX *iv = channelisedData + ant1 * s * 2 + t;
        const COMPLEX *jv = channelisedData + ant2 * s * 2 + t;

        COMPLEX u1 = iv[0];
        COMPLEX v1 = jv[0];
        COMPLEX u2 = iv[s];
        COMPLEX v2 = jv[s];
        hipComplex a1;
        hipComplex a2;
        hipComplex a3;
        hipComplex a4;
        a1.x = (u1.x * v1.x + u1.y * v1.y);
        a1.y = u1.y * v1.x - u1.x * v1.y;
        a2.x = u1.x * v2.x + u1.y * v2.y;
        a2.y = u1.y * v2.x - u1.x * v2.y;
        a3.x = u2.x * v1.x + u2.y * v1.y;
        a3.y = u2.y * v1.x - u2.x * v1.y;
        a4.x = u2.x * v2.x + u2.y * v2.y;
        a4.y = u2.y * v2.x - u2.x * v2.y;

        for (int k = *fftsamples; k < s; k += *fftsamples) {
            u1 = iv[k];
            v1 = jv[k];
            u2 = iv[k + s];
            v2 = jv[k + s];

            a1.x += HALF2FLOAT(u1.x * v1.x + u1.y * v1.y);
            a1.y += HALF2FLOAT(u1.y * v1.x - u1.x * v1.y);
            a2.x += HALF2FLOAT(u1.x * v2.x + u1.y * v2.y);
            a2.y += HALF2FLOAT(u1.y * v2.x - u1.x * v2.y);
            a3.x += HALF2FLOAT(u2.x * v1.x + u2.y * v1.y);
            a3.y += HALF2FLOAT(u2.y * v1.x - u2.x * v1.y);
            a4.x += HALF2FLOAT(u2.x * v2.x + u2.y * v2.y);
            a4.y += HALF2FLOAT(u2.y * v2.x - u2.x * v2.y);
        }

        a1.x /= *numffts;
        a1.y /= *numffts;
        a2.x /= *numffts;
        a2.y /= *numffts;
        a3.x /= *numffts;
        a3.y /= *numffts;
        a4.x /= *numffts;
        a4.y /= *numffts;
        baselineData[4 * b * NCHAN + t] = a1;
        baselineData[(4 * b + 1) * NCHAN + t] = a2;
        baselineData[(4 * b + 2) * NCHAN + t] = a3;
        baselineData[(4 * b + 3) * NCHAN + t] = a4;
    }
}

// Function to compute the phase angle of a hipComplex number
float cuCargf(hipComplex z) {
    return atan2f(hipCimagf(z), hipCrealf(z));
}

void
saveVisibilities(int NBASELINE, int NCHAN, int PARAACCUM, COMPLEX *baselineData, int *nbaselines, int *numchannels,
                 double *bandwidth) {
    COMPLEX **vis;
    FILE *fvis = fopen("vis.out", "w");
    if (fvis == NULL) {
        fprintf(stderr, "Error opening file %s\n", "vis.out");
        exit(1);
    }

    // Copy final visibilities back to CPU
    vis = (hipComplex **) malloc(*nbaselines * 4 * sizeof(hipComplex *));
    for (int i = 0; i < *nbaselines * 4; i++) {
        vis[i] = (hipComplex *) malloc(NCHAN * sizeof(hipComplex));
        gpuErrchk(
                hipMemcpy(vis[i], &baselineData[i * *numchannels], NCHAN * sizeof(hipComplex), hipMemcpyDeviceToHost));
    }

    printf("Test : %e \n", vis[0][0].x);
    printf("Test : %e \n", vis[0][0].y);

    for (int c = 0; c < NCHAN; c++) {
        fprintf(fvis, "%5d %11.6f", c, (c + 0.5) / NCHAN * *bandwidth / 1e6);
        for (int i = 0; i < *nbaselines * 4; i++) {
            fprintf(fvis, " %11.6f %11.6f %11.6f %10.6f",
                    hipCrealf(vis[i][c]), hipCimagf(vis[i][c]),
                    hipCabsf(vis[i][c]), cuCargf(vis[i][c]));
        }
        fprintf(fvis, "\n");
    }
    fclose(fvis);

    for (int i = 0; i < *nbaselines * 4; i++) {
        free(vis[i]);
    }
    free(vis);
}

void calculateDelaysAndPhases(int NANT, int NFFTS, double *delays, double *lo, double *sampletime, int *fftsamples,
                            int *numchannels, int *samplegranularity, float *rotationPhaseInfo, int *sampleShifts,
                            float *fractionalSampleDelays) {
    for (size_t ifft = 0; ifft < NFFTS; ++ifft) {
        for (size_t iant = 0; iant < NANT; ++iant) {
            double meandelay, deltadelay, netdelaysamples_f, startphase;
            double d0, d1, d2, a, b;
            double *interpolator = &(delays[iant * 4]);
            double filestartoffset = delays[iant * 4 + 3];
            float fractionaldelay;
            int netdelaysamples;

            // Calculate values at the beginning, middle, and end of this FFT
            d0 = interpolator[0] * ifft * ifft + interpolator[1] * ifft + interpolator[2];
            d1 = interpolator[0] * (ifft + 0.5) * (ifft + 0.5) + interpolator[1] * (ifft + 0.5) + interpolator[2];
            d2 = interpolator[0] * (ifft + 1.0) * (ifft + 1.0) + interpolator[1] * (ifft + 1.0) + interpolator[2];

            // Use these to calculate a linear interpolator across the FFT, as well as a mean value
            a = d2 - d0; // This is the delay gradient across this FFT
            b = d0 + (d1 - (a * 0.5 + d0)) / 3.0; // This is the delay at the start of the FFT
            meandelay = a * 0.5 + b; // This is the delay in the middle of the FFT
            deltadelay = a / *fftsamples; // This is the change in delay per sample across this FFT window

            netdelaysamples_f = (meandelay - filestartoffset) / *sampletime;
            netdelaysamples = (int) (netdelaysamples_f / *samplegranularity) * *samplegranularity;

            // Save the integer number of sample shifts
            sampleShifts[iant * NFFTS + ifft] = netdelaysamples;

            // Save the fractional delay
            fractionaldelay = (float) (-(netdelaysamples_f - netdelaysamples) * 2 * M_PI /
                                       *fftsamples);  // Radians per FFT channel
            fractionalSampleDelays[iant * NFFTS + ifft] = fractionaldelay;

            // Set the fringe rotation phase for the first sample of a given FFT of a given antenna
            startphase = b * *lo;
            rotationPhaseInfo[iant * NFFTS * 2 + ifft * 2] = (float) (startphase - (int) startphase) * 2 * M_PI;
            rotationPhaseInfo[iant * NFFTS * 2 + ifft * 2 + 1] = (float) (deltadelay * *lo) * 2 * M_PI;
        }
    }
}

__global__ void channCorrect(int SUBINTSAMP, int NANT, int NPOL, COMPLEX *  chann_fft, COMPLEX *  chann_frac, COMPLEX *  chann_out) {
        int global_index = blockIdx.x * blockDim.x + threadIdx.x;
        if (global_index < 15728640) {
            if (chann_frac[global_index].x == 0.0f || chann_frac[global_index].y == 0.0f) {
                //memcpy(&correct_out[global_index], &fft_out[global_index], sizeof(hipComplex));
                chann_out[global_index] = chann_fft[global_index];
            } else {
                //memcpy(&correct_out[global_index], &frac_out[global_index], sizeof(hipComplex));
                chann_out[global_index] = chann_frac[global_index];
            }
        }
}